//#include<algorithm>
//#include<cstdlib>
//#include<ctime>
#ifndef UTILS_H_
#define UTILS_H_

#include<armadillo>
#include<vector>
#include<string>
#include<sstream>
#include<iomanip>
#include<hipblas.h>
//#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
using namespace std;
using namespace arma;

typedef double elem_type;

typedef struct{
    int width;
    int height;
    elem_type *elements;
}matrix;

template <typename T>
void split(string s,vector<T> &vec)
{ // splits the given string at whitespaces and stores them in the vector provided

	stringstream ss(s);
	T val;
	while(ss>>val)
	{
		vec.push_back(val);
	}
}

int no_of_lines(const char *fname)
{ // Return the number of lines in the file given
	ifstream fh(fname);
	string line;
	int nLines;
	nLines=0;
	if(fh.is_open())
		while (getline(fh, line))
			++nLines;
	fh.close();
	return nLines;
}

template <typename T>
void print_vec(vector<T> &vec)
{
	for(int i=0;i<vec.size();i++)
	{
		cout<<vec[i]<<" ";
	}
	cout<<endl;
}

template<typename T>
string list_tostring(T listValues,int nValues)
{ //convert a array of values into a string
	stringstream ss;
	for(int i=0;i<nValues;i++)
	{
		ss<<listValues[i]<<" ";
	}
	ss<<endl;
	return ss.str();
}

template<typename T>
void str_to_vec(string s,vector<T> &vec)
{ //convert a array of values into a string
	stringstream ss(s);
	T val;
	while(ss>>val)
		vec.push_back(val);
}

template<typename T>
void str_to_type(string str,T &val)
{
	stringstream ss(str);
	ss>>val;
}


//void read_params(const char *paramsFname)
//{
//	ifstream fh(paramsFname);
//	string line;
//	vector<int> temp;
//	for(int lineNo=1;getline(fh,line);lineNo++)
//	{
//		switch(lineNo)
//		{
//		case 1:
//			str_to_vec(line,unitsInLayer);
//			break;
//		case 2:
//			str_to_vec(line,outFnType);
//			break;
//		case 3:
//			str_to_vec(line,temp);
//			batchSize = temp[0];
//			if(temp.size()==2)
//				batchesPerEpoch = temp[1];
//			else
//				batchesPerEpoch = 0;
//			break;
//		case 4:
//			str_to_type(line,eta);
//			break;
//		}
//	}
//	nLayers = outFnType.size();
////	cout<<"no.of layers: "<<nLayers<<endl;
//}

int ReadData(const char *fname,mat &Data)
{	// Reads the data from the file given as input into a matrix and returns the matrix

	ifstream fh(fname);
	int nPatterns,nFeatures;
	string line;
	vector<double> pattern;
//	cout<<"Reading data from file: "<<fname<<endl;
	if(fh.is_open())
	{
		nPatterns = no_of_lines(fname);
		getline(fh,line);
		str_to_vec(line,pattern);
		nFeatures = pattern.size();
//		cout<<"no of patterns: "<<nPatterns<<endl;
//		cout<<"no of features per pattern: "<<nFeatures<<endl;
//		Data.set_size(nPatterns,nFeatures);
		Data.zeros(nFeatures,nPatterns);
		vector<double> linef; // a vector of floats representing a row of input patterns
		str_to_vec(line,linef);
		for(int j=0;j<nFeatures;j++)
			Data(j,0) = linef[j];
		for(int i=1;getline(fh,line);i++)
		{
			linef.clear();
//			cout<<line<<endl;
			split(line,linef);
			for(int j=0;j<nFeatures;j++)
				Data(j,i) = linef[j];
		}
		fh.close();
	}
	return nPatterns;
}


int myReadData(const char *fname,matrix &Data)
{	// Reads the data from the file given as input into a matrix and returns the matrix

	ifstream fh(fname);
	int nPatterns,nFeatures,colIdx,rowIdx;
	string line;
	vector<elem_type> pattern;
	//	cout<<"Reading data from file: "<<fname<<endl;
	if(fh.is_open())
	{
		nPatterns = no_of_lines(fname);
		getline(fh,line);
		str_to_vec(line,pattern);
		nFeatures = pattern.size();
		Data.width = nPatterns;
		Data.height = nFeatures;
		//		cout<<"no of patterns: "<<nPatterns<<endl;
		//		cout<<"no of features per pattern: "<<nFeatures<<endl;
		//		Data.set_size(nPatterns,nFeatures);
		Data.elements = new elem_type[nFeatures*nPatterns];
		vector<elem_type> linef; // a vector of floats representing a row of input patterns
		str_to_vec(line,linef);
		rowIdx = 0;
		colIdx = 0;
		for(int j=0;j<nFeatures;j++)
		{
             		Data.elements[rowIdx*nPatterns + colIdx] = linef[j];
			rowIdx++;
		}
		colIdx++;
		for(int i=1;getline(fh,line);i++)
		{
			rowIdx = 0;
			linef.clear();
			//			cout<<line<<endl;
			split(line,linef);
			for(int j=0;j<nFeatures;j++){
				Data.elements[rowIdx*nPatterns + colIdx] = linef[j];
				rowIdx++;
			}
			colIdx++;
		}
		fh.close();
	}
	return nPatterns;
}

void print_matrix(matrix &m)
{
    for(int i = 0;i<m.height;i++)
    {
        for(int j=0;j<m.width;j++)
            cout<<fixed<<showpoint<<setprecision(4)<<m.elements[i*m.width + j]<<" ";
        cout<<endl;
    }
}

void insert_onesrow(Mat<elem_type> &A)
{
    Row<elem_type> onesRow;
    onesRow.ones(A.n_cols);
    A.insert_rows(A.n_rows,onesRow);
}

void preprocess_data(Mat<elem_type> &A)
{	//mean subtraction and variance normalization of data
	Col<elem_type> meanA,varA;
//	A.print("A:");
	meanA = mean(A,1);
//	meanA.print("meanA:");
	varA = var(A,0,1);
//	varA.print("varA:");
	A.each_col() -= meanA;
//	A.print("meanSubA:");
	A.each_col() /= varA;
//	A.print("varNormA:");
}

template<typename T>
int matmul(const T *A,hipblasOperation_t transA,const T *B,hipblasOperation_t transB,
           int HA,int WA,int WB,double alpha,double beta,T *hC)
{ //performs matrix-matrix mutliplication using cublas gemm() function.
  //dim(A): HA x WA , dim(B): WA x WB
    
    hipError_t cudaStat;    
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    T *dA,*dB,*dC;
//    double *alpha,*beta;
//    alpha = (double*)malloc(sizeof(double));
//    beta = (double*)malloc(sizeof(double));
//    *alpha = 1;
//    *beta = 0;
//    hC = (double *)malloc(HA*WB*sizeof(double));
//    Mat<T> *C = new Mat<T>(HA,WB,fill::zeros);
//    hC = C->memptr();
    
    //Allocate memory on device(GPU) for A,B,C
    cudaStat = hipMalloc((void**)&dA,HA*WA*sizeof(T));
    if (cudaStat != hipSuccess) { 
    printf ("device memory allocation failed");
    return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc((void**)&dB,WA*WB*sizeof(T));
    if (cudaStat != hipSuccess) { 
    printf ("device memory allocation failed");
    return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc((void**)&dC,HA*WB*sizeof(T));
    if (cudaStat != hipSuccess) { 
    printf ("device memory allocation failed");
    return EXIT_FAILURE;
    }
    
//  create a cuda handle
    stat = hipblasCreate(&handle); 
    if (stat != HIPBLAS_STATUS_SUCCESS) {
     printf ("CUBLAS initialization failed\n"); 
     return EXIT_FAILURE;
    }
    
    stat = hipblasSetMatrix(HA,WA,sizeof(T),A,HA,dA,HA);
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
    printf ("set matrix failed");
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
    }
    
    stat = hipblasSetMatrix(WA,WB,sizeof(T),B,WA,dB,WA);
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
    printf ("set matrix failed");
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
    }

//  call the cublas matrix-matrix mulitplication kernel    
    stat = hipblasDgemm(handle,transA,transB,HA,WB,WA,&alpha,dA,HA,dB,WA,&beta,dC,HA);
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
    printf ("matrix-matrix(hipblasDgemm) mulitiplication failed");
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
    }

// copy the prodcut matrix back to host memory
    stat = hipblasGetMatrix(HA,WB,sizeof(T),dC,HA,hC,HA);
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
    printf ("fetching data from device memory failed");
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);   
    return EXIT_SUCCESS;
}

template<typename T>
int matadd(const T *A,const T *B,int HA,int WA,double alpha,double beta,T *C)
{
    hipError_t cudaStat;    
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    T *dA,*dB,*dC;
//    double *alpha,*beta;
//    alpha = (double*)malloc(sizeof(double));
//    beta = (double*)malloc(sizeof(double));
//    *alpha = 1;
//    *beta = 1;
    //allocate memory for the matrices A,B,C on device
    cudaStat = hipMalloc((void**)&dA,HA*WA*sizeof(T));
    if (cudaStat != hipSuccess) { 
    printf ("device memory allocation failed");
    return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc((void**)&dB,HA*WA*sizeof(T));
    if (cudaStat != hipSuccess) { 
    printf ("device memory allocation failed");
    return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc((void**)&dC,HA*WA*sizeof(T));
    if (cudaStat != hipSuccess) { 
    printf ("device memory allocation failed");
    return EXIT_FAILURE;
    }
    
    //create a cuda handle
    stat = hipblasCreate(&handle); 
    if (stat != HIPBLAS_STATUS_SUCCESS) {
     printf ("CUBLAS initialization failed\n"); 
     return EXIT_FAILURE;
    }
    
    //copy the matrices A and B from host memory to device memory    
    stat = hipblasSetMatrix(HA,WA,sizeof(T),A,HA,dA,HA);
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
    printf ("set matrix failed");
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
    }
    
    stat = hipblasSetMatrix(HA,WA,sizeof(T),B,HA,dB,HA);
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
    printf ("set matrix failed");
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
    }
    
    //call the cublas matrix-matrix addition function
    stat = hipblasDgeam(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,HA,WA,&alpha,dA,HA,&beta,dB,HA,dC,HA);
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
    printf ("matrix-matrix(hipblasDgeam) addition failed");
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
    }

// copy the prodcut matrix back to host memory
    stat = hipblasGetMatrix(HA,WA,sizeof(T),dC,HA,C,HA);
    if (stat != HIPBLAS_STATUS_SUCCESS) { 
    printf ("fetching data from device memory failed");
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipblasDestroy(handle);   
    return EXIT_SUCCESS;

}
#endif


